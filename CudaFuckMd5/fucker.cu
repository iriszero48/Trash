#include "hip/hip_runtime.h"
﻿#include "fucker.h"

#include <cassert>
#include <stdexcept>
#include <sstream>

#include "hip/hip_runtime.h"
#include ""

enum class Round
{
    F,
    G,
    H,
    I
};

#define _Error(...)\
	std::runtime_error((std::ostringstream{} << "[" << __FILE__ << ":" << __LINE__ << "] [" << __FUNCTION__ << "] " << __VA_ARGS__).str())


template <typename T>
class CudaArray
{
    T* data = nullptr;

public:
    explicit CudaArray(const size_t size)
    {
        CudaMalloc((T**)&data, sizeof(T) * size);
    }

    explicit CudaArray(const T* data, const size_t size)
    {
        CudaMalloc(&this->data, size);
        const auto err = hipMemcpy(this->data, data, size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
            throw _Error("memcpy: " << hipGetErrorString(err));
    }

    [[nodiscard]] T* Get() const
    {
        return data;
    }

    ~CudaArray()
    {
        hipFree(data);
    }

private:
    void CudaMalloc(T** devPtr, const size_t size) const
    {
        const auto err = hipMalloc(devPtr, size);
        if (err != hipSuccess)
            throw _Error("malloc: " << hipGetErrorString(err));
    }
};


__device__ constexpr std::uint32_t F(const std::uint32_t x, const std::uint32_t y, const std::uint32_t z)
{
    return z ^ x & (y ^ z);
}

__device__ constexpr std::uint32_t G(const std::uint32_t x, const std::uint32_t y, const std::uint32_t z)
{
    return y ^ z & (x ^ y);
}

__device__ constexpr std::uint32_t H(const std::uint32_t x, const std::uint32_t y, const std::uint32_t z)
{
    return x ^ y ^ z;
}

__device__ constexpr std::uint32_t I(const std::uint32_t x, const std::uint32_t y, const std::uint32_t z)
{
    return y ^ (x | ~z);
}

template <Round Func>
__device__ void Step(std::uint32_t& a, const std::uint32_t b, const std::uint32_t c, const std::uint32_t d, const std::uint32_t x, const std::uint32_t t, const std::uint32_t s)
{
    if constexpr (Func == Round::F)
        a += F(b, c, d) + x + t;
    if constexpr (Func == Round::G)
        a += G(b, c, d) + x + t;
    if constexpr (Func == Round::H)
        a += H(b, c, d) + x + t;
    if constexpr (Func == Round::I)
        a += I(b, c, d) + x + t;
    a = a << s | (a >> (32u - s));
    a += b;
}

__device__ std::uint32_t Get(const std::uint8_t* buf, const std::uint64_t index)
{
	return *(std::uint32_t*)&buf[index * 4];
}

__device__ void FuckingMd5ARound(const uint8_t* key, const size_t keySize, char res[32])
{
    uint8_t buf[64]{ 0 };
    for (size_t i = 0; i < keySize; ++i) buf[i] = key[i];

    buf[keySize] = 0x80;
    *reinterpret_cast<uint64_t*>(&buf[64 - 8]) = keySize * 8;

    uint32_t a = 0x67452301, b = 0xefcdab89, c = 0x98badcfe, d = 0x10325476;

    const uint32_t savedA = a;
    const uint32_t savedB = b;
    const uint32_t savedC = c;
    const uint32_t savedD = d;

    Step<Round::F>(a, b, c, d, Get(buf, 0), 0xd76aa478u, 7);
    Step<Round::F>(d, a, b, c, Get(buf, 1), 0xe8c7b756u, 12);
    Step<Round::F>(c, d, a, b, Get(buf, 2), 0x242070dbu, 17);
    Step<Round::F>(b, c, d, a, Get(buf, 3), 0xc1bdceeeu, 22);
    Step<Round::F>(a, b, c, d, Get(buf, 4), 0xf57c0fafu, 7);
    Step<Round::F>(d, a, b, c, Get(buf, 5), 0x4787c62au, 12);
    Step<Round::F>(c, d, a, b, Get(buf, 6), 0xa8304613u, 17);
    Step<Round::F>(b, c, d, a, Get(buf, 7), 0xfd469501u, 22);
    Step<Round::F>(a, b, c, d, Get(buf, 8), 0x698098d8u, 7);
    Step<Round::F>(d, a, b, c, Get(buf, 9), 0x8b44f7afu, 12);
    Step<Round::F>(c, d, a, b, Get(buf, 10), 0xffff5bb1u, 17);
    Step<Round::F>(b, c, d, a, Get(buf, 11), 0x895cd7beu, 22);
    Step<Round::F>(a, b, c, d, Get(buf, 12), 0x6b901122u, 7);
    Step<Round::F>(d, a, b, c, Get(buf, 13), 0xfd987193u, 12);
    Step<Round::F>(c, d, a, b, Get(buf, 14), 0xa679438eu, 17);
    Step<Round::F>(b, c, d, a, Get(buf, 15), 0x49b40821u, 22);

    Step<Round::G>(a, b, c, d, Get(buf, 1), 0xf61e2562u, 5);
    Step<Round::G>(d, a, b, c, Get(buf, 6), 0xc040b340u, 9);
    Step<Round::G>(c, d, a, b, Get(buf, 11), 0x265e5a51u, 14);
    Step<Round::G>(b, c, d, a, Get(buf, 0), 0xe9b6c7aau, 20);
    Step<Round::G>(a, b, c, d, Get(buf, 5), 0xd62f105du, 5);
    Step<Round::G>(d, a, b, c, Get(buf, 10), 0x02441453u, 9);
    Step<Round::G>(c, d, a, b, Get(buf, 15), 0xd8a1e681u, 14);
    Step<Round::G>(b, c, d, a, Get(buf, 4), 0xe7d3fbc8u, 20);
    Step<Round::G>(a, b, c, d, Get(buf, 9), 0x21e1cde6u, 5);
    Step<Round::G>(d, a, b, c, Get(buf, 14), 0xc33707d6u, 9);
    Step<Round::G>(c, d, a, b, Get(buf, 3), 0xf4d50d87u, 14);
    Step<Round::G>(b, c, d, a, Get(buf, 8), 0x455a14edu, 20);
    Step<Round::G>(a, b, c, d, Get(buf, 13), 0xa9e3e905u, 5);
    Step<Round::G>(d, a, b, c, Get(buf, 2), 0xfcefa3f8u, 9);
    Step<Round::G>(c, d, a, b, Get(buf, 7), 0x676f02d9u, 14);
    Step<Round::G>(b, c, d, a, Get(buf, 12), 0x8d2a4c8au, 20);

    Step<Round::H>(a, b, c, d, Get(buf, 5), 0xfffa3942u, 4);
    Step<Round::H>(d, a, b, c, Get(buf, 8), 0x8771f681u, 11);
    Step<Round::H>(c, d, a, b, Get(buf, 11), 0x6d9d6122u, 16);
    Step<Round::H>(b, c, d, a, Get(buf, 14), 0xfde5380cu, 23);
    Step<Round::H>(a, b, c, d, Get(buf, 1), 0xa4beea44u, 4);
    Step<Round::H>(d, a, b, c, Get(buf, 4), 0x4bdecfa9u, 11);
    Step<Round::H>(c, d, a, b, Get(buf, 7), 0xf6bb4b60u, 16);
    Step<Round::H>(b, c, d, a, Get(buf, 10), 0xbebfbc70u, 23);
    Step<Round::H>(a, b, c, d, Get(buf, 13), 0x289b7ec6u, 4);
    Step<Round::H>(d, a, b, c, Get(buf, 0), 0xeaa127fau, 11);
    Step<Round::H>(c, d, a, b, Get(buf, 3), 0xd4ef3085u, 16);
    Step<Round::H>(b, c, d, a, Get(buf, 6), 0x04881d05u, 23);
    Step<Round::H>(a, b, c, d, Get(buf, 9), 0xd9d4d039u, 4);
    Step<Round::H>(d, a, b, c, Get(buf, 12), 0xe6db99e5u, 11);
    Step<Round::H>(c, d, a, b, Get(buf, 15), 0x1fa27cf8u, 16);
    Step<Round::H>(b, c, d, a, Get(buf, 2), 0xc4ac5665u, 23);

    Step<Round::I>(a, b, c, d, Get(buf, 0), 0xf4292244u, 6);
    Step<Round::I>(d, a, b, c, Get(buf, 7), 0x432aff97u, 10);
    Step<Round::I>(c, d, a, b, Get(buf, 14), 0xab9423a7u, 15);
    Step<Round::I>(b, c, d, a, Get(buf, 5), 0xfc93a039u, 21);
    Step<Round::I>(a, b, c, d, Get(buf, 12), 0x655b59c3u, 6);
    Step<Round::I>(d, a, b, c, Get(buf, 3), 0x8f0ccc92u, 10);
    Step<Round::I>(c, d, a, b, Get(buf, 10), 0xffeff47du, 15);
    Step<Round::I>(b, c, d, a, Get(buf, 1), 0x85845dd1u, 21);
    Step<Round::I>(a, b, c, d, Get(buf, 8), 0x6fa87e4fu, 6);
    Step<Round::I>(d, a, b, c, Get(buf, 15), 0xfe2ce6e0u, 10);
    Step<Round::I>(c, d, a, b, Get(buf, 6), 0xa3014314u, 15);
    Step<Round::I>(b, c, d, a, Get(buf, 13), 0x4e0811a1u, 21);
    Step<Round::I>(a, b, c, d, Get(buf, 4), 0xf7537e82u, 6);
    Step<Round::I>(d, a, b, c, Get(buf, 11), 0xbd3af235u, 10);
    Step<Round::I>(c, d, a, b, Get(buf, 2), 0x2ad7d2bbu, 15);
    Step<Round::I>(b, c, d, a, Get(buf, 9), 0xeb86d391u, 21);

    a += savedA;
    b += savedB;
    c += savedC;
    d += savedD;

    auto ptr = res;

#define ToHex(x) ((x) > 9 ? (x) + 55 : (x) + 48)
#define VToHex(v) \
    *ptr++ = ToHex(static_cast<uint8_t>(v) >> 4);\
    *ptr++ = ToHex(static_cast<uint8_t>(v) % 16)
#define Uint32ToHex(u32)\
	VToHex(((uint8_t*)&u32)[0]);\
    VToHex(((uint8_t*)&u32)[1]);\
    VToHex(((uint8_t*)&u32)[2]);\
    VToHex(((uint8_t*)&u32)[3])
    Uint32ToHex(a);
    Uint32ToHex(b);
    Uint32ToHex(c);
    Uint32ToHex(d);
}

__device__ void AddBaseN(uint8_t* c, const uint8_t* const a, const uint8_t* const b, const size_t n, const uint8_t base)
{
    int16_t carry = 0;
    for (int64_t i = n - 1; i >= 0; i--)
    {
        int16_t curr = carry + a[i] + b[i];
        carry = curr / base;
        curr %= base;
        c[i] = curr;
    }
    if (carry > 0)
        assert(("AddBaseN", false));
}

__device__ void AddUint64BaseN(uint8_t* c, const uint8_t* const a, uint64_t b, const size_t n, const uint8_t base, uint8_t* nb)
{
    for (int64_t i = n - 1; i >= 0; --i)
    {
        nb[i] = b % base;
        b /= base;
    }
    if (b > 0)
        assert(("AddUint64BaseN", false));

    AddBaseN(c, a, nb, n, base);
}

__global__ void FuckingCudaCall(
    const char* prefix, const size_t prefixSize,
    const char* suffix, const size_t suffixSize,
    const size_t dynamic,
    const char* alphabet, const size_t alphabetSize,
    const size_t offset,
    char* out,
    uint8_t* aBuf, uint8_t* bBuf, uint8_t* cBuf, const uint64_t maxId)
{
    const uint64_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= maxId) return;

    const auto keySize = prefixSize + suffixSize + dynamic;
    const auto elemSize = keySize + 1 + 32 + 1;

    uint8_t* a = aBuf + id * dynamic;
    for (size_t i = 0; i < dynamic; ++i) a[i] = 0;

    const uint64_t b = offset + id;

    uint8_t* c = cBuf + id * dynamic;

    AddUint64BaseN(c, a, b, dynamic, alphabetSize, bBuf + dynamic * id);

    char* key = out + elemSize * id;
    for (size_t i = 0; i < prefixSize; ++i) key[i] = prefix[i];
    for (size_t i = 0; i < dynamic; ++i) key[prefixSize + i] = alphabet[c[i]];
    for (size_t i = 0; i < suffixSize; ++i) key[prefixSize + dynamic + i] = suffix[i];
    key[keySize] = ' ';

    FuckingMd5ARound((const uint8_t*)key, keySize, out + id * elemSize + keySize + 1);

    key[elemSize - 1] = '\n';
}

CudaRes FuckingCall(
    const StringView& prefix, const StringView& suffix,
    const size_t dynamic,
    const StringView& alphabet,
    const size_t offset, const size_t num)
{
    CudaArray<char> prefixCuda(prefix.Data, prefix.Size);
    CudaArray<char> suffixCuda(suffix.Data, suffix.Size);
    CudaArray<char> alphabetCuda(alphabet.Data, alphabet.Size);

    CudaArray<uint8_t> aBuf(num * dynamic);
    CudaArray<uint8_t> bBuf(num * dynamic);
    CudaArray<uint8_t> cBuf(num * dynamic);

    const auto outSize = num * (prefix.Size + suffix.Size + dynamic + 1 + 32 + 1);

	char* outBuf = nullptr;
    auto err = hipMallocManaged(&outBuf, outSize);
    if (err != hipSuccess) throw _Error("malloc managed: " << hipGetErrorString(err));

    FuckingCudaCall<<<num / 200 + (num % 200 == 0 ? 0 : 1), 200>>>(
        prefixCuda.Get(), prefix.Size, 
        suffixCuda.Get(), suffix.Size,
        dynamic,
        alphabetCuda.Get(), alphabet.Size,
        offset, outBuf,
        aBuf.Get(), bBuf.Get(), cBuf.Get(), num);

	err = hipGetLastError();
    if (err != hipSuccess) throw _Error("FuckingCudaCall: " << hipGetErrorString(err));

    err = hipDeviceSynchronize();
    if (err != hipSuccess) throw _Error("Synchronize: " << hipGetErrorString(err));

    return { outBuf, outSize };
}

void FuckingFree(CudaRes& sv)
{
    const auto err = hipFree(sv.Data);
    if (err != hipSuccess) throw _Error("free: " << hipGetErrorString(err));
    sv = {};
}
