#include "hip/hip_runtime.h"
#include "Cuda.h"

#include <stdexcept>

#include "hip/hip_runtime.h"
#include ""



__device__ void AddBaseN(char* c, const char* const a, const char* const b, const uint64_t n, const uint64_t base)
{
    int carry = 0;
    for (int i = n - 1; i >= 0; i--)
    {
        int curr = carry + a[i] + b[i];
        carry = curr / base;
        curr %= base;
        c[i] = curr;
    }
    //if (carry > 0); //Impossible!
}

__device__ void AddUint64BaseN(char* c, const char* const a, uint64_t b, const uint64_t n, const uint64_t base)
{
    char* nb;
    hipMalloc((void**)&nb, n * sizeof(char));
    for (int i = n - 1; i >= 0; --i)
    {
        nb[i] = b % base;
        b /= base;
    }
    nb[0] = b;
    AddBaseN(c, a, nb, n, base);
    hipFree(nb);
}

__device__ uint32_t crc32Cuda(const uint32_t crc, const uint8_t val, const uint32_t* crc_32_tab)
{
    return crc_32_tab[static_cast<uint8_t>(crc) ^ val] ^ crc >> 8;
}

constexpr uint64_t Rate = 1;

__global__ void FuckZipCrc(
    uint8_t* res,
    const uint8_t* fileHead,
    const char* passwordIndex,
    const uint64_t passwordLen,
    const char* alphabet,
    const uint64_t alphabetLen,
    const uint64_t maxId,
    const uint8_t* mult_tab,
    const uint32_t* crc_32_tab)
{
    const uint64_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= maxId) return;
    for (int loop = 0; loop < Rate; ++loop)
    {
        const auto readId = id * Rate + loop;
        //const auto readId = id;
        char* pw;
        hipMalloc((void**)&pw, passwordLen * sizeof(char));
        AddUint64BaseN(pw, passwordIndex, readId, passwordLen, alphabetLen);
        for (int i = 0; i < passwordLen; ++i)
        {
            pw[i] = alphabet[pw[i]];
        }
        uint32_t key0 = 0x12345678UL;
        uint32_t key1 = 0x23456789UL;
        uint32_t key2 = 0x34567890UL;
        for (int i = 0; i < passwordLen; ++i)
        {
            key0 = crc32Cuda(key0, pw[i], crc_32_tab);
            key1 = (key1 + static_cast<uint8_t>(key0)) * 134775813 + 1;
            key2 = crc32Cuda(key2, key1 >> 24, crc_32_tab);
        }
        for (int i = 0; i < 11; ++i)
        {
            const uint8_t preTarget = fileHead[i] ^ mult_tab[static_cast<uint16_t>(key2) >> 2];
            key0 = crc32Cuda(key0, preTarget, crc_32_tab);
            key1 = (key1 + static_cast<uint8_t>(key0)) * 134775813 + 1;
            key2 = crc32Cuda(key2, key1 >> 24, crc_32_tab);
        }
        const uint8_t target = fileHead[11] ^ mult_tab[static_cast<uint16_t>(key2) >> 2];
        res[readId] = target == fileHead[12];
        hipFree(pw);
    }
}

__constant__ uint8_t mult_tab[16384];
__constant__ uint32_t crc_32_tab[256];

void FuckZipCrcCuda(uint8_t* flags, const uint8_t* fileHead, const std::string& password, const std::string& alphabet, const uint64_t chunkSize)
{
    hipError_t error;

    uint8_t* resCuda;
    uint8_t* fileHeadCuda;

    if (static bool init = false; !init)
    {
        uint8_t tab[16384];
        for (auto t = 0; t < 16384; t++) tab[t] = ((t * 4 + 3) * (t * 4 + 2) >> 8) & 0xff;
        error = hipMemcpyToSymbol(HIP_SYMBOL(mult_tab), tab, sizeof(tab));
        if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: tab: hipMemcpy: ") + hipGetErrorString(error));

        uint32_t crcTab[256] = { 0 };
        for (uint32_t i = 0; i < 256; ++i)
        {
            auto checksum = i;
            for (auto j = 0; j < 8; ++j) checksum = (checksum >> 1) ^ (checksum & 0x1 ? 0xEDB88320 : 0);
            crcTab[i] = checksum;
        }
        error = hipMemcpyToSymbol(HIP_SYMBOL(crc_32_tab), crcTab, sizeof(crcTab));
        if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: crcTab: hipMemcpy: ") + hipGetErrorString(error));

        error = hipMalloc(reinterpret_cast<void**>(&resCuda), chunkSize * sizeof(uint8_t));
        if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: resCuda: hipMalloc: ") + hipGetErrorString(error));

        error = hipMalloc(reinterpret_cast<void**>(&fileHeadCuda), 14 * sizeof(uint8_t));
        if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: fileHeadCuda: hipMalloc: ") + hipGetErrorString(error));
        error = hipMemcpy(fileHeadCuda, fileHead, 14 * sizeof(uint8_t), hipMemcpyHostToDevice);
        if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: fileHeadCuda: hipMemcpy: ") + hipGetErrorString(error));

        init = true;
    }

    auto pwIndex = password;
    for (auto i = 0; i < password.length(); ++i) pwIndex[i] = alphabet.find(pwIndex[i]);
    char* pwIndexCuda;
    error = hipMalloc(reinterpret_cast<void**>(&pwIndexCuda), password.length() * sizeof(char));
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: pwIndexCuda: hipMalloc: ") + hipGetErrorString(error));
    error = hipMemcpy(pwIndexCuda, pwIndex.data(), pwIndex.length() * sizeof(char), hipMemcpyHostToDevice);
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: pwIndexCuda: hipMemcpy: ") + hipGetErrorString(error));

    char* alphabetCuda;
    error = hipMalloc(reinterpret_cast<void**>(&alphabetCuda), alphabet.length() * sizeof(char));
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: alphabetCuda: hipMalloc: ") + hipGetErrorString(error));
    error = hipMemcpy(alphabetCuda, alphabet.data(), alphabet.length() * sizeof(char), hipMemcpyHostToDevice);
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: alphabetCuda: hipMemcpy: ") + hipGetErrorString(error));

    uint8_t* a;
    uint32_t* b;
    hipGetSymbolAddress((void**)&a, mult_tab);
    hipGetSymbolAddress((void**)&b, crc_32_tab);
	
    FuckZipCrc<<<chunkSize / Rate / 1000 + 1, 1000>>>(
        resCuda,
        fileHeadCuda,
        pwIndexCuda, pwIndex.length(),
        alphabetCuda, alphabet.length(),
        chunkSize / Rate,
        a, b);
    error = hipGetLastError();
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: FuckZipCrc: ") + hipGetErrorString(error));

    error = hipDeviceSynchronize();
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: hipDeviceSynchronize: ") + hipGetErrorString(error));
	
    error = hipMemcpy(flags, resCuda, chunkSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (error != hipSuccess) throw std::runtime_error(std::string("FuckZipCrc: res: hipMemcpy: ") + hipGetErrorString(error));
    
    //hipFree(resCuda);
    //hipFree(fileHeadCuda);
    hipFree(pwIndexCuda);
    hipFree(alphabetCuda);

    //hipFree(tab);
    //hipFree(crc_32_tab);
}
