#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#define v 0
#define l 1
#define r 2

__host__ void Preorder(void*** nodes, void(*fun)(void*))
{
	int id = 0;
	printf("Preorder %d:", id);
	int* top = NULL;
	void*** stack = NULL;
	top = (int*)malloc(sizeof(int));
	stack = (void***)malloc(100 * sizeof(void**));
	*top = -1;
	stack[++*top] = nodes[id];
	while (*top > -1)
	{
		stack[99] = stack[(*top)--];
		fun(stack[99][v]);
		if (stack[99][r]) stack[++*top] = (void**)stack[99][r];
		if (stack[99][l]) stack[++*top] = (void**)stack[99][l];
	}
}

__host__ void Inorder(void*** nodes, void(*fun)(void*))
{
	int id = 0;
	printf("Preorder %d:", id);
	int* top = NULL;
	void*** stack = NULL;
	top = (int*)malloc(sizeof(int));
	stack = (void***)malloc(100 * sizeof(void***));
	*top = -1;
	for (stack[99] = nodes[id]; *top > -1 || stack[99]; stack[99] = (void**)stack[99][r])
	{
		while(stack[99])
		{
			stack[++*top] = stack[99];
			stack[99] = (void**)stack[99][l];
		}
		stack[99] = stack[(*top)--];
		fun(stack[99][v]);
	}
}

__host__ void Postorder(void*** nodes, void(*fun)(void*))
{
	int id = 0;
	printf("Preorder %d:", id);
	int* top = NULL;
	void*** stack = NULL;
	top = (int*)malloc(sizeof(int));
	stack = (void***)malloc(100 * sizeof(void***));
	*top = -1;
	stack[99] = nodes[id];
	while (*top > -1 || stack[99])
	{
		while (stack[99])
		{
			stack[++*top] = stack[99];
			stack[99] = (void**)stack[99][l];
		}
		if (stack[*top][r] && stack[*top][r] != (void*)stack[98]) stack[99] = (void**)stack[*top][r];
		else
		{
			fun(stack[*top][v]);
			stack[98] = stack[(*top)--];
		}
	}
}

__host__ void** Tree(void* value, void* left, void* right)
{
	void** node = NULL;
	node = (void**)malloc(3 * sizeof(void*));
	node[v] = value;
	node[l] = left;
	node[r] = right;
	return node;
}

__host__ void callCuda()
{
	int n[9];
	for (auto i = 0; i < 9; ++i)n[i] = i + 1;
	void*** nodes = NULL;
	nodes = (void***)malloc(5 * sizeof(void***));
	for (int i = 0; i < 5; i++)
	{
		nodes[i] = Tree((void*)&n[0],
			Tree((void*)&n[1],
				Tree((void*)&n[3],
					Tree((void*)&n[6], NULL, NULL),
					NULL),
				Tree((void*)&n[4], NULL, NULL)),
			Tree((void*)&n[2],
				Tree((void*)&n[5],
					Tree((void*)&n[7], NULL, NULL),
					Tree((void*)&n[8], NULL, NULL)),
				NULL));
	}
	auto fun = [](void* n) { std::cout << *(int*)n << " "; };
	Preorder(nodes, fun);
	std::cout << std::endl;
	Inorder(nodes, fun);
	std::cout << std::endl;
	Postorder(nodes, fun);
}

int main()
{
	callCuda();
    return 0;
}
